#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_xrotate4(CudaImg image)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= image.m_size.x)
        return;

    if (y >= image.m_size.y)
        return;

    image.at<uchar4>(x, y) = image.at<uchar4>((image.m_size.x - 1 - x), y);
}

void cu_xrotate4(CudaImg image)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((image.m_size.x + thread_count) / thread_count, (image.m_size.y + thread_count) / thread_count);

    kernel_xrotate4<<<block_count, block_size>>>(image);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_insert(CudaImg bg_image, CudaImg fg_image, int2 pos)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= fg_image.m_size.x)
        return;

    if (y >= fg_image.m_size.y)
        return;

    if (pos.x + x < 0 || pos.x + x >= bg_image.m_size.x)
        return;

    if (pos.y + y < 0 || pos.y + y >= bg_image.m_size.y)
        return;

    bg_image.at<uchar3>(pos.x + x, pos.y + y).x = (fg_image.at<uchar4>(x, y).x * fg_image.at<uchar4>(x, y).w / 255.0) + bg_image.at<uchar3>(pos.x + x, pos.y + y).x * (255 - fg_image.at<uchar4>(x, y).w) / 255.0;
    bg_image.at<uchar3>(pos.x + x, pos.y + y).y = (fg_image.at<uchar4>(x, y).y * fg_image.at<uchar4>(x, y).w / 255.0) + bg_image.at<uchar3>(pos.x + x, pos.y + y).y * (255 - fg_image.at<uchar4>(x, y).w) / 255.0;
    bg_image.at<uchar3>(pos.x + x, pos.y + y).z = (fg_image.at<uchar4>(x, y).z * fg_image.at<uchar4>(x, y).w / 255.0) + bg_image.at<uchar3>(pos.x + x, pos.y + y).z * (255 - fg_image.at<uchar4>(x, y).w) / 255.0;
}

void cu_insert(CudaImg bg_image, CudaImg fg_image, int2 pos)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((fg_image.m_size.x + thread_count) / thread_count, (fg_image.m_size.y + thread_count) / thread_count);

    kernel_insert<<<block_count, block_size>>>(bg_image, fg_image, pos);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

template <typename T>
__global__ void kernel_copy(CudaImg image, CudaImg copy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= image.m_size.x)
        return;

    if (y >= image.m_size.y)
        return;

    copy.at<T>(x, y) = image.at<T>(x, y);
}

void cu_copy3(CudaImg image, CudaImg copy)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((image.m_size.x + thread_count) / thread_count, (image.m_size.y + thread_count) / thread_count);

    kernel_copy<uchar3><<<block_count, block_size>>>(image, copy);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_copyAndResize(CudaImg image, CudaImg copy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= image.m_size.x)
        return;

    if (y >= image.m_size.y)
        return;

    float2 ratio = {(float)image.m_size.x / (float)copy.m_size.x, (float)image.m_size.y / (float)copy.m_size.y};

    copy.at<uchar4>(x, y) = image.at<uchar4>(x * ratio.x, y * ratio.y);
}

void cu_copyAndResize(CudaImg image, CudaImg copy)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((copy.m_size.x + thread_count) / thread_count, (copy.m_size.y + thread_count) / thread_count);

    kernel_copyAndResize<<<block_count, block_size>>>(image, copy);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_blur(CudaImg image, int kernel_size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= image.m_size.x)
        return;

    if (y >= image.m_size.y)
        return;

    int border = kernel_size / 2;
    float kernel_area = (float)(kernel_size * kernel_size);
    float3 sum = make_float3(0.0f, 0.0f, 0.0f);

    for (int ky = -border; ky <= border; ky++)
    {
        for (int kx = -border; kx <= border; kx++)
        {
            int px = min(max(x + kx, 0), image.m_size.x - 1);
            int py = min(max(y + ky, 0), image.m_size.y - 1);
            sum.y += image.at<uchar4>(px, py).y;
            sum.x += image.at<uchar4>(px, py).x;
            sum.z += image.at<uchar4>(px, py).z;
        }
    }

    image.at<uchar4>(x, y).x = sum.x / kernel_area;
    image.at<uchar4>(x, y).y = sum.y / kernel_area;
    image.at<uchar4>(x, y).z = sum.z / kernel_area;
}

void cu_blur(CudaImg image, int kernel_size)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((image.m_size.x + thread_count) / thread_count, (image.m_size.y + thread_count) / thread_count);

    kernel_blur<<<block_count, block_size>>>(image, kernel_size);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_rotate4(CudaImg image, CudaImg copy, double degree)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= image.m_size.x)
        return;

    if (y >= image.m_size.y)
        return;

    double2 center = {image.m_size.x / 2.0, image.m_size.y / 2.0};

    double2 rotation[] = {{cos(degree), -sin(degree)},
                          {sin(degree), cos(degree)}};

    double2 new_pos = {0, 0};

    new_pos.x = (x - center.x) * rotation[0].x + (y - center.y) * rotation[0].y;
    new_pos.y = (x - center.x) * rotation[1].x + (y - center.y) * rotation[1].y;
    int new_x = (int)(new_pos.x + center.x + 0.5);
    int new_y = (int)(new_pos.y + center.y + 0.5);

    if (new_x < 0 || new_x >= image.m_size.x)
        return;

    if (new_y < 0 || new_y >= image.m_size.y)
        return;

    copy.at<uchar4>((int)x, (int)y) = image.at<uchar4>(new_x, new_y);
}

void cu_rotate4(CudaImg image, CudaImg copy, double degree)
{
    hipError_t l_cerr;

    int thread_count = 32;
    dim3 block_size(thread_count, thread_count);
    dim3 block_count((image.m_size.x + thread_count) / thread_count, (image.m_size.y + thread_count) / thread_count);

    double rad = (degree * M_PI) / 180;

    kernel_rotate4<<<block_count, block_size>>>(image, copy, rad);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}